#include "hip/hip_runtime.h"
#include "conv.cuh"
#include <cuda/barrier>
#include <mma.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

namespace cg = cooperative_groups;


#define a(_n, _x, _y, _c) a[(_n) * H * W * C + (_x) * W * C + (_y) * C + (_c)]
#define w(_k, _x, _y, _c) w[(_k) * R * S * C + (_x) * S * C + (_y) * C + (_c)]
#define b(_n, _x, _y, _k) b[(_n) * H * W * K + (_x) * W * K + (_y) * K + (_k)]
#define LENGTH (C)
#define FLEN (16)
#define KLEN (16)
#define IML (128)
#define STRIDE (FLEN+8)
#define a_shared(_x, _y, _c) a_shared[(_x) * (BLOCK + S / 2 * 2) * (LENGTH+4) + (_y) * (LENGTH+4) + (_c)]
#define w_shared(_x, _y, _c) w_shared[(_x) * S * (LENGTH+4) + (_y) * (LENGTH+4) + (_c)]
// #define a_buf(_x, _y, _c) a_buf[(_x) * (BLOCK + S / 2 * 2) * (LENGTH+4) + (_y) * (LENGTH+4) + (_c)]
// #define w_buf(_x, _y, _c) w_buf[(_x) * S * (LENGTH+4) + (_y) * (LENGTH+4) + (_c)]
#define a_buf(_x, _y) a_buf[(_x)*(STRIDE)+(_y)]
#define w_buf(_x, _y) w_buf[(_x)*(STRIDE)+(_y)]
using namespace nvcuda;




static constexpr int BLOCK = 16;
__device__ int zero[LENGTH] = {0};
__device__ __half zerof[FLEN];
__device__ int8_t off[3]={-1,0,1};

// INT8
template <>
KernelConfig get_kernel_config<int8_t>() {
    KernelConfig config;
    config.grid = dim3((H + BLOCK - 1) / BLOCK, (W + BLOCK - 1) / BLOCK);
    config.block = dim3(BLOCK, BLOCK);
    config.shared_memory_size = 0;  // Use default shared memory size
    return config;
}
/*
template<typename T>
__device__ __forceinline__
void fetch(const T* __restrict__ a, 
            T* __restrict__ a_shared, 
            const T* __restrict__ w, 
            T* __restrict__ w_shared,
            int n, int c, int k, int i, int j, 
            cuda::barrier<cuda::thread_scope_block> &bar) {
    if (i < size && j < size) {
        cuda::memcpy_async(&a_shared(threadIdx.x+1, threadIdx.y+1, 0), &a(n, i, j, c), LENGTH*sizeof(T), bar);
    }
    else {
        cuda::memcpy_async(&a_shared(threadIdx.x+1, threadIdx.y+1, 0), zero, LENGTH*sizeof(T), bar);
    }
    if(threadIdx.x < R + 3 && threadIdx.y < S + 3 && threadIdx.x >= 3 && threadIdx.y >= 3) {
        cuda::memcpy_async(&w_shared(threadIdx.x - 3, threadIdx.y - 3, 0), &w(k, threadIdx.x - 3, threadIdx.y - 3, c), LENGTH*sizeof(T), bar);
    }
    switch(threadIdx.x){
        case 0:
            if(i - 1 >= 0 && j >= 0 && j < W) cuda::memcpy_async(&a_shared(0, threadIdx.y + 1, 0), &a(n, i - 1, j, c), LENGTH*sizeof(T), bar);
            else cuda::memcpy_async(&a_shared(0, threadIdx.y + 1, 0), zero, LENGTH*sizeof(T), bar);
            break;
        case 15:
            if(i + 1 < H && j >= 0 && j < W) cuda::memcpy_async(&a_shared(BLOCK + 1, threadIdx.y + 1, 0), &a(n, i + 1, j, c), LENGTH*sizeof(T), bar);
            else cuda::memcpy_async(&a_shared(BLOCK + 1, threadIdx.y + 1, 0), zero, LENGTH*sizeof(T), bar);
            break;
    }
    switch(threadIdx.y){
        case 0:
            if(j - 1 >= 0 && i >= 0 && i < H) cuda::memcpy_async(&a_shared(threadIdx.x + 1, 0, 0), &a(n, i, j - 1, c), LENGTH*sizeof(T), bar);
            else cuda::memcpy_async(&a_shared(threadIdx.x + 1, 0, 0), zero, LENGTH*sizeof(T), bar);
            break;
        case 15:
            if(j + 1 < W && i >= 0 && i < H) cuda::memcpy_async(&a_shared(threadIdx.x + 1, BLOCK + 1, 0), &a(n, i, j + 1, c), LENGTH*sizeof(T), bar);
            else cuda::memcpy_async(&a_shared(threadIdx.x + 1, BLOCK + 1, 0), zero, LENGTH*sizeof(T), bar);
            break;
    }
    if(threadIdx.x < 12 && threadIdx.y < 12 && threadIdx.x >= 10 && threadIdx.y >= 10) {
        if(i + (BLOCK+1) * (static_cast<int>(threadIdx.x) - 10) >= threadIdx.x + 1 && 
                i + (BLOCK+1) * (static_cast<int>(threadIdx.x) - 10) < H + threadIdx.x + 1 &&
                j + (BLOCK+1) * (static_cast<int>(threadIdx.y) - 10) >= threadIdx.y + 1 &&
                j + (BLOCK+1) * (static_cast<int>(threadIdx.y) - 10) < W + threadIdx.y + 1) 
            cuda::memcpy_async(&a_shared((threadIdx.x - 10) * (BLOCK + 1), (threadIdx.y - 10) * (BLOCK + 1), 0), 
                               &a(n, i - threadIdx.x - 1 + (BLOCK + 1) * (threadIdx.x - 10), 
                                   j - threadIdx.y - 1 + (BLOCK + 1) * (threadIdx.y - 10), c), LENGTH*sizeof(T), bar);
        else cuda::memcpy_async(&a_shared((threadIdx.x - 10) * (BLOCK + 1), (threadIdx.y - 10) * (BLOCK + 1), 0), zero, LENGTH*sizeof(T), bar);
    }
}
*/
/*
template<typename T>
__device__ __forceinline__
void cpy(T* __restrict__ dst, 
         const T* __restrict__ src, 
         int len) {
    #pragma unroll
    for(int i = 0; i < len; ++i) {
        dst[i] =  __ldg(src + i);
    }
}

template<typename T>
__device__ __forceinline__
void fetch(const T* __restrict__ a, 
            T* __restrict__ a_shared, 
            const T* __restrict__ w, 
            T* __restrict__ w_shared,
            int n, int c, int k, int i, int j) {
    if (i < size && j < size) {
        cpy(&a_shared(threadIdx.x+1, threadIdx.y+1, 0), &a(n, i, j, c), LENGTH);
    }
    else {
        cpy(&a_shared(threadIdx.x+1, threadIdx.y+1, 0), zero, LENGTH);
    }
    if(threadIdx.x < R + 3 && threadIdx.y < S + 3 && threadIdx.x >= 3 && threadIdx.y >= 3) {
        cpy(&w_shared(threadIdx.x - 3, threadIdx.y - 3, 0), &w(k, threadIdx.x - 3, threadIdx.y - 3, c), LENGTH);
    }
    switch(threadIdx.x){
        case 0:
            if(i - 1 >= 0 && j >= 0 && j < W) cpy(&a_shared(0, threadIdx.y + 1, 0), &a(n, i - 1, j, c), LENGTH);
            else cpy(&a_shared(0, threadIdx.y + 1, 0), zero, LENGTH);
            break;
        case 15:
            if(i + 1 < H && j >= 0 && j < W) cpy(&a_shared(BLOCK + 1, threadIdx.y + 1, 0), &a(n, i + 1, j, c), LENGTH);
            else cpy(&a_shared(BLOCK + 1, threadIdx.y + 1, 0), zero, LENGTH);
            break;
    }
    switch(threadIdx.y){
        case 0:
            if(j - 1 >= 0 && i >= 0 && i < H) cpy(&a_shared(threadIdx.x + 1, 0, 0), &a(n, i, j - 1, c), LENGTH);
            else cpy(&a_shared(threadIdx.x + 1, 0, 0), zero, LENGTH);
            break;
        case 15:
            if(j + 1 < W && i >= 0 && i < H) cpy(&a_shared(threadIdx.x + 1, BLOCK + 1, 0), &a(n, i, j + 1, c), LENGTH);
            else cpy(&a_shared(threadIdx.x + 1, BLOCK + 1, 0), zero, LENGTH);
            break;
    }
    if(threadIdx.x < 12 && threadIdx.y < 12 && threadIdx.x >= 10 && threadIdx.y >= 10) {
        if(i + (BLOCK+1) * (static_cast<int>(threadIdx.x) - 10) >= threadIdx.x + 1 && 
                i + (BLOCK+1) * (static_cast<int>(threadIdx.x) - 10) < H + threadIdx.x + 1 &&
                j + (BLOCK+1) * (static_cast<int>(threadIdx.y) - 10) >= threadIdx.y + 1 &&
                j + (BLOCK+1) * (static_cast<int>(threadIdx.y) - 10) < W + threadIdx.y + 1) 
            cpy(&a_shared((threadIdx.x - 10) * (BLOCK + 1), (threadIdx.y - 10) * (BLOCK + 1), 0), 
                &a(n, i - threadIdx.x - 1 + (BLOCK + 1) * (threadIdx.x - 10), 
                    j - threadIdx.y - 1 + (BLOCK + 1) * (threadIdx.y - 10), c), LENGTH);
        else cpy(&a_shared((threadIdx.x - 10) * (BLOCK + 1), (threadIdx.y - 10) * (BLOCK + 1), 0), zero, LENGTH);
    }
}
*/

template<typename T>
__device__ __forceinline__
void fetch(const T* __restrict__ a, 
            T* __restrict__ a_shared, 
            int n) {
    int idx = threadIdx.y / 2 * 2, wpid = (threadIdx.y*blockDim.x+threadIdx.x)%32;
    int i = blockIdx.x * blockDim.x, j = blockIdx.y * blockDim.y;
    switch(idx){
        case 0:
            for(int y = 0; y < 3; y++){
                for(int x = 0; x < BLOCK+2; x++){
                    if(i+x-1<H&&i+x-1>=0&&idx+j+y-1<W&&idx+j+y-1>=0) 
                        *(int*)&a_shared(x, idx+y, wpid*4) = *(int*)&a(n, i + x - 1, idx + j + y - 1, wpid*4);
                    else 
                        *(int*)&a_shared(x, idx+y, wpid*4) = zero[wpid];
                }
            }
            break;
        case 14:
            for(int y = 0; y < 3; y++){
                for(int x = 0; x < BLOCK+2; x++){
                    if(i+x-1<H&&i+x-1>=0&&idx+j+y<W&&idx+j+y>=0) 
                        *(int*)&a_shared(x, idx+y+1, wpid*4) = *(int*)&a(n, i + x - 1, idx + j + y, wpid*4);
                    else 
                        *(int*)&a_shared(x, idx+y+1, wpid*4) = zero[wpid];
                }
            }
            break;
        default:
            for(int y = 0; y < 2; y++){
                for(int x = 0; x < BLOCK+2; x++){
                    if(i+x-1<H&&i+x-1>=0&&idx+j+y<W&&idx+j+y>=0) 
                        *(int*)&a_shared(x, idx+y+1, wpid*4) = *(int*)&a(n, i + x - 1, idx + j + y, wpid*4);
                    else    
                        *(int*)&a_shared(x, idx+y+1, wpid*4) = zero[wpid];
                }

            }
            //*(int*)&w_shared(x/R, x%R, coeff*32+wpid*4) = *(int*)&w(k, x/R, x%R, c + wpid*4);
            break;
    }
}

template<typename T>
__device__ __forceinline__
void fetchfil(const T* __restrict__ w, 
            T* __restrict__ w_shared,
            int k){
    int idx = threadIdx.y / 2, wpid = (threadIdx.y*blockDim.x+threadIdx.x)%32;
    switch(idx){
        case 0:
            *(int*)&w_shared(0, idx, wpid*4) = *(int*)&w(k,0, idx, wpid*4);
            *(int*)&w_shared(0, 8, wpid*4) = *(int*)&w(k, 0, 8, wpid*4);
            break;
        default:
            *(int*)&w_shared(0, idx, wpid*4) = *(int*)&w(k, 0, idx, wpid*4);
            break;
    }
}

template <>
__global__ void conv2d_cuda_kernel<int8_t, int>(const int8_t *__restrict__ a, const int8_t *__restrict__ w,
                                                int8_t *__restrict__ b) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    alignas(LENGTH) __shared__ int8_t a_shared[(BLOCK + R / 2 * 2) * (BLOCK + S / 2 * 2) * (LENGTH+4)]/*, 
                        a_next[(BLOCK + R / 2 * 2) * (BLOCK + S / 2 * 2) * (LENGTH+4)]*/;
    alignas(LENGTH) __shared__ int8_t w_shared[R * S * (LENGTH+4)]/*, w_next[R*S*(LENGTH+4)]*/;
    // __shared__ cuda::barrier<cuda::thread_scope_block> bar;
    // int8_t* __restrict__ a_buf=a_shared, *w_buf=w_shared, *aa=a_next, *ww= w_next, *t=NULL;
    // if (threadIdx.x == 0 && threadIdx.y == 0) {
    //     init(&bar, blockDim.x * blockDim.y);
    // }
    // __syncthreads();
    // fetch<int8_t>(a, a_buf, w, w_buf,0, 0, 0, i, j, bar);
    // bar.arrive_and_wait();
    //if (i < size && j < size) {
//     hipSharedMemConfig config;
// hipDeviceGetSharedMemConfig(&config);
// if (config == hipSharedMemBankSizeFourByte) {
//   printf("Bank width = 4 bytes\n");
// } else if (config == hipSharedMemBankSizeEightByte) {
//   printf("Bank width = 8 bytes\n");
// } else {
//   printf("Unknown shared mem config\n");
// }

    for (int n = 0; n < N; ++n) {
        fetch<int8_t>(a, a_shared, n);
        for (int k = 0; k < K; ++k) {
            int result = 0;
                /*int x = i - R / 2, y = j - S / 2;
                for (int r = 0; r < R; ++r) {
                    for (int s = 0; s < S; ++s) {
                        if (!(x < 0 || x >= size || y < 0 || y >= size)) {
                            result += static_cast<int>(a(n, x, y, c)) * static_cast<int>(w(k, r, s, c));
                        }
                        y++;
                    }
                    x++;
                    y -= S;
                }*/
            // switch(c){
            //     case C - LENGTH:
            //         switch(k){
            //             case K-1:
            //                 if(n!=N-1) fetch<int8_t>(a, aa, w, ww, n+1, 0, 0, i, j, bar);
            //                 break;
            //             default:
            //                 fetch<int8_t>(a, aa, w, ww, n, 0, k+1, i, j, bar);
            //                 break;
            //         }
            //         break;
            //     default:
            //         fetch<int8_t>(a, aa, w, ww, n, c+LENGTH, k, i, j, bar);
            //         break;
            // }

            //if((a_buf(threadIdx.x+1,threadIdx.y+1, c)!=a(n,i,j,c)&&a_buf(threadIdx.x+1,threadIdx.y+1,c))) printf("Error at %d %d\n",i,j);
            // #pragma unroll
            // for(int x = 0; x < LENGTH/32; x++){
            fetchfil<int8_t>(w, w_shared,k);
            __syncthreads();
            if (i < size && j < size){
                for(int ct = 0; ct < LENGTH; ct += 4){
                    int x = threadIdx.x, y = threadIdx.y;
                    // if(x+1<BLOCK+2&&y+1<BLOCK+2&&a_shared(x+1,y+1,ct)!=a(n,i,j,ct)) printf("Error at %d %d %d, %d %d\n",i,j,ct,a_shared(x+1,y+1,ct), a(n,i,j,ct));
                    // else if(x+1>=BLOCK+2||y+1>=BLOCK+2) printf("Illegal at %d %d %d\n",x,y,BLOCK+2);
                    for (int r = 0; r < R; ++r) {
                        for (int s = 0; s < S; ++s) {
                            if (!(x < 0 || x >= BLOCK + R / 2 * 2 || y < 0 || y >= BLOCK + S / 2 * 2)) {
                                //result += static_cast<int>(a_buf(x, y, ct)) * static_cast<int>(w_buf(r, s, ct));
                                // result = __dp4a(*((int*)&a_buf(x, y, ct)), *((int*)&w_buf(r, s, ct)), result);
                                result = __dp4a(*((int*)&a_shared(x, y, ct)), *((int*)&w_shared(r, s, ct)), result);
                            }
                            y++;
                        }
                        x++;
                        y -= S;
                    }
                }
                
            }
            // bar.arrive_and_wait();
            
            // t= a_buf;
            // a_buf = aa;
            // aa = t;
            // t = w_buf;
            // w_buf = ww;
            // ww = t;
            __syncthreads();
        
            // if(&b(n,i,j,k)-b==1228800) printf("Error at %d %d,value is %d\n",i,j,static_cast<int8_t>(result));
            if(i<size&&j<size) b(n, i, j, k) = static_cast<int8_t>(result);
        }
    }
    //}
    /*else{
        for (int n = 0; n < N; ++n) {
            for (int c = 0; c < C; ++c) {
                a_shared(threadIdx.x+1, threadIdx.y+1) = 0;
            }
        }
    }*/
}

// HALF
template <>
KernelConfig get_kernel_config<half_t>() {
    KernelConfig config;
    config.grid = dim3((H + BLOCK - 1) / BLOCK, (W + BLOCK - 1) / BLOCK);
    config.block = dim3(BLOCK, BLOCK);
    config.shared_memory_size = 0;  // Use default shared memory size
    return config;
}

__device__ __forceinline__
void tile(const half_t* __restrict__ a, 
    __half* __restrict__ a_buf, 
    int n, int c, int ii, int im,
    cuda::barrier<cuda::thread_scope_block> &bar)
{
    int widx = threadIdx.y / 2;
    int i = blockIdx.x * blockDim.x+widx/(BLOCK/(IML/8))+im, j = blockIdx.y * blockDim.y+widx%(BLOCK/(IML/8))*IML/8;
    // int i = blockIdx.x * blockDim.x + widx/(BLOCK/(IML/8)), j = blockIdx.y * blockDim.y;
    for(int x=0;x<IML/8;x++){
        if(i+off[ii/3]>=0&&i+off[ii/3]<H&&j+x+off[ii%3]>=0&&j+x+off[ii%3]<W)
            cuda::memcpy_async(&a_buf(widx*IML/8+x,0), &a(n,i+off[ii/3],j+x+off[ii%3],c), FLEN*sizeof(__half), bar);
        else cuda::memcpy_async(&a_buf(widx*IML/8+x,0), zerof, FLEN*sizeof(__half), bar);
    }
}

__device__ __forceinline__
void filter(const half_t* __restrict__ w,
    __half* __restrict__ w_buf,
    int k, int c, int ii,
    cuda::barrier<cuda::thread_scope_block> &bar)
{
    int widx = threadIdx.y / 2;
    for(int x=0;x<KLEN/8;x++){
        cuda::memcpy_async(&w_buf(widx*KLEN/8+x,0),&w(k+widx*KLEN/8+x,0,ii,c),FLEN*sizeof(__half),bar);
    }
}

__device__ __forceinline__
void tile_cg(
    const half_t* __restrict__ a,
    __half* __restrict__ a_buf,
    int n, int c, int ii, int im,
    cg::thread_block_tile<32> &cg_block)
{
    int widx = threadIdx.y / 2;
    int i = blockIdx.x * blockDim.x+widx/(BLOCK/(IML/8))+im, j = blockIdx.y * blockDim.y+widx%(BLOCK/(IML/8))*IML/8;
    // int i = blockIdx.x * blockDim.x + widx/(BLOCK/(IML/8)), j = blockIdx.y * blockDim.y;
    __half* src=NULL;
    // if(widx!=cg_block.meta_group_rank()) printf("Error\n");
    // printf("warp %d lane %d => i=%d j=%d\n",
    //    wgrp.meta_group_rank(),
    //    wgrp.thread_rank(),
    //    i, j);

    for(int x=0;x<IML/8;x++){
        if(i+off[ii/3]>=0&&i+off[ii/3]<H&&j+x+off[ii%3]>=0&&j+x+off[ii%3]<W){
            src=(__half*)&a(n,i+off[ii/3],j+x+off[ii%3],c);
            // printf("Src at a");
        }
        else src=zerof;
        cg::memcpy_async(cg_block,&a_buf(widx*IML/8+x,0), src, FLEN);
        cg::wait(cg_block);
    }
}

// filter: load filter weights into shared buffer using cooperative memcpy_async
__device__ __forceinline__
void filter_cg(
    const half_t* __restrict__ w,
    __half* __restrict__ w_buf,
    int k, int c, int ii,
    cg::thread_block_tile<32> &cg_block)
{
    int widx = threadIdx.y / 2;
    for (int x = 0; x < KLEN/8; x++) {
        cg::memcpy_async(
            cg_block,
            &w_buf(widx * (KLEN/8) + x, 0),
            (__half*)&w(k + widx * (KLEN/8) + x, 0, ii, c),
            FLEN
        );
        cg::wait(cg_block);
    }
}


template <>
__global__ void conv2d_cuda_kernel<half_t, float>(const half_t *__restrict__ a, const half_t *__restrict__ w,
                                                  half_t *__restrict__ b) {
    const int i = blockIdx.x * blockDim.x;
    const int j = blockIdx.y * blockDim.y;
    const int widx = threadIdx.y / 2;
    alignas(16) __shared__ __half tmp[8][16*16];
    alignas(128) __shared__ __half a1[IML*(STRIDE)],a2[IML*(STRIDE)],w1[KLEN*(STRIDE)],w2[KLEN*(STRIDE)];
    __shared__ cuda::barrier<cuda::thread_scope_block> bar;
    cg::thread_block tb = cg::this_thread_block();
    cg::thread_block_tile<32> wgrp = cg::tiled_partition<32>(tb);
    __half *a_buf=a1, *aa=a2, *w_buf=w1, *ww=w2,*t;
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> c_frag;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        init(&bar, blockDim.x * blockDim.y);
    }
    __syncthreads();
    tile(a,a_buf,0,0,0,0,bar);
    filter(w,w_buf,0,0,0,bar);
    // tile_cg(a,a_buf,0,0,0,0,wgrp);
    // filter_cg(w,w_buf,0,0,0,wgrp);
    zerof[threadIdx.x]= __float2half(0.0f);
    // cg::wait(wgrp);
    
    bar.arrive_and_wait();
    for(int n = 0; n < N; n++){
        for(int im = 0; im < BLOCK; im += IML/BLOCK){
            for(int k = 0 ;k < K; k += KLEN){
                wmma::fill_fragment(c_frag, 0.0f);
                
                for(int c = 0; c < C; c += FLEN){
                    for(int ii=0; ii < 9; ii++){
                        // printf("%d %d %d %d %d\n",n,im,k,c,ii);
                        switch(ii){
                            case 8:
                                switch(c){
                                    case C-FLEN:
                                        if(k!=K-KLEN) {
                                            filter(w,ww,k+KLEN,0,0,bar);
                                            tile(a,aa,n,0,0,im,bar);
                                        }
                                        else{
                                            if(im!=BLOCK-IML/BLOCK){
                                                tile(a,aa,n,0,0,im+IML/BLOCK,bar);
                                                filter(w,ww,0,0,0,bar);
                                            }
                                            else{
                                                if(n!=N-1){
                                                    tile(a,aa,n+1,0,0,0,bar);
                                                    filter(w,ww,0,0,0,bar);
                                                }
                                            }
                                        }
                                        break;
                                    default:
                                        tile(a,aa,n,c+FLEN,0,im,bar);
                                        filter(w,ww,k,c+FLEN,0,bar);
                                        break;
                                }
                                break;
                            default:
                                tile(a,aa,n,c,ii+1,im,bar);
                                
                                filter(w,ww,k,c,ii+1,bar);
                                
                                break;
                        }
                        // switch(ii){
                        //     case 8:
                        //         switch(c){
                        //             case C-FLEN:
                        //                 if(k!=K-KLEN) {
                        //                     filter_cg(w,ww,k+KLEN,0,0,wgrp);
                        //                     tile_cg(a,aa,n,0,0,im,wgrp);
                        //                 }
                        //                 else{
                        //                     if(im!=BLOCK-IML/BLOCK){
                        //                         tile_cg(a,aa,n,0,0,im+IML/BLOCK,wgrp);
                        //                         filter_cg(w,ww,0,0,0,wgrp);
                        //                     }
                        //                     else{
                        //                         if(n!=N-1){
                        //                             tile_cg(a,aa,n+1,0,0,0,wgrp);
                        //                             filter_cg(w,ww,0,0,0,wgrp);
                        //                         }
                        //                     }
                        //                 }
                        //                 break;
                        //             default:
                        //                 tile_cg(a,aa,n,c+FLEN,0,im,wgrp);
                        //                 filter_cg(w,ww,k,c+FLEN,0,wgrp);
                        //                 break;
                        //         }
                        //         break;
                        //     default:
                        //         tile_cg(a,aa,n,c,ii+1,im,wgrp);
                                
                        //         filter_cg(w,ww,k,c,ii+1,wgrp);
                                
                        //         break;
                        // }
                        
                        // if(i+im+off[ii/3]>=0&&i+im+off[ii/3]<H&&
                        //     j+0+off[ii%3]>=0&&j+0+off[ii%3]<W&&
                        //     a_buf(0,0)!=(__half)a(n,i+im+off[ii/3],j+0+off[ii%3],c)){
                        //     printf("Error at %d %d %d %d %d %d, %f!=%f\n",i,j,n,im,c,ii,float(a_buf(0,0)),float(a(n,i+im+off[ii/3],j+0+off[ii%3],c)));
                        // }
                        // printf("Once down\n");
                        // int id= threadIdx.y*blockDim.x+threadIdx.x;
                        // if(!ii&&!n&&!im&&!c&&i+im+id%128/16+off[ii/3]>=0&&i+im+id%128/16+off[ii/3]<H&&
                        //     j+id%128%16+off[ii%3]>=0&&j+id%128%16+off[ii%3]<W&&
                        //     a_buf(id%128,0)!=(__half)a(n,i+im+id%128/16+off[ii/3],j+id%128%16+off[ii%3],c))
                        //         printf("Error at %d %d %d %d %d %d %d, %f!=%f\n",i,j,n,im,c,ii,id%128,float(a_buf(id%128,0)),float(a(n,i+im+id%128/16+off[ii/3],j+id%128%16+off[ii%3],c)));
                        // if(w_buf(threadIdx.x%16,threadIdx.y%16)!=(__half)w(k+threadIdx.x%16,ii/3,ii%3,c+threadIdx.y%16))
                        //     printf("Error at %d %d %d %d %d %d %d, %f!=%f\n",i,j,n,im,c,ii,k,float(w_buf(threadIdx.x%16,threadIdx.y%16)),float(w(k+threadIdx.x%16,ii/3,ii%4,c+threadIdx.y%16)));
                        for(int ct=0;ct<FLEN;ct+=16){
                            wmma::load_matrix_sync(a_frag,&a_buf(widx*16,ct),STRIDE);
                            // printf("Once down\n");
                            wmma::load_matrix_sync(b_frag,&w_buf(0,ct),STRIDE);
                            wmma::mma_sync(c_frag,a_frag,b_frag,c_frag);
                        }
                        
                        // cg::wait(wgrp);
                        bar.arrive_and_wait();
                        // printf("Once down\n");
                        
                        t = a_buf;
                        a_buf = aa; 
                        aa = t;
                        t = w_buf;
                        w_buf = ww;
                        ww = t;
                    }
                }
                if(i+im+widx<H) {
                    if(j+KLEN<W) wmma::store_matrix_sync((__half*)&b(n,i+im+widx,j,k),c_frag,K,wmma::mem_row_major);
                    else{
                        wmma::store_matrix_sync(tmp[widx],c_frag,16,wmma::mem_row_major);
                        cuda::memcpy_async((__half*)&b(n,i+im+widx,j,k),&tmp[widx][0],sizeof(__half)*KLEN,bar);
                        cuda::memcpy_async((__half*)&b(n,i+im+widx,j+1,k),&tmp[widx][1*16],sizeof(__half)*KLEN,bar);
                        cuda::memcpy_async((__half*)&b(n,i+im+widx,j+2,k),&tmp[widx][2*16],sizeof(__half)*KLEN,bar);
                        cuda::memcpy_async((__half*)&b(n,i+im+widx,j+3,k),&tmp[widx][3*16],sizeof(__half)*KLEN,bar);
                    }
                }
                
            }
        }
    }
    // bar.arrive_and_wait();
}